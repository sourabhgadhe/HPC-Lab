#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void reduce_min_max_sum(int* input, int* output, int size) {
  extern __shared__ int sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  // Load input elements into shared memory
  sdata[tid] = (i < size) ? input[i] : 0;
  __syncthreads();
  // Parallel reduction in shared memory
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] = min(sdata[tid], sdata[tid + s]); // Min operation
      sdata[tid] = max(sdata[tid], sdata[tid + s]); // Max operation
      sdata[tid] += sdata[tid + s]; // Sum operation
    }
    __syncthreads();
  }
  // Write the reduced value to output
  if (tid == 0)
    output[blockIdx.x] = sdata[0];
}

int main() {
  int size = 1024; // Size of the input array
  int block_size = 256;
  int num_blocks = (size + block_size - 1) / block_size;
  // Generate input data
  int* input = new int[size];
  for (int i = 0; i < size; ++i) {
    input[i] = i + 1;
  }
  // Allocate device memory
  int* d_input;
  int* d_output;
  hipMalloc((void**)&d_input, size * sizeof(int));
  hipMalloc((void**)&d_output, num_blocks * sizeof(int));
  // Copy input data to device memory
  hipMemcpy(d_input, input, size * sizeof(int), hipMemcpyHostToDevice);
  // Launch kernel for parallel reduction
  reduce_min_max_sum<<<num_blocks, block_size, block_size * sizeof(int)>>>(d_input, d_output, size);
  // Allocate host memory for output
  int* output = new int[num_blocks];
  // Copy output data from device memory to host memory
  hipMemcpy(output, d_output, num_blocks * sizeof(int), hipMemcpyDeviceToHost);
  // Perform reduction on host
  int min_val = output[0];
  int max_val = output[0];
  int sum_val = output[0];
  for (int i = 1; i < num_blocks; ++i) {
    min_val = min(min_val, output[i]); // Minimum value
    max_val = max(max_val, output[i]); // Maximum value
    sum_val += output[i]; // Sum value
  }

  printf("Min: %d\n", min_val);
  printf("Max: %d\n", max_val);
  printf("Sum: %d\n", sum_val);

  // Free device memory
  hipFree(d_input);
  hipFree(d_output);
  // Free host memory
  delete[] input;
  delete[] output;
  return 0;
}