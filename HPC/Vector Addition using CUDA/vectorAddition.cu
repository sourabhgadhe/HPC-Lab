#include "hip/hip_runtime.h"
#include <stdio.h>
#define SIZE 50000

__global__ void p_vecAdd(int *a, int *b, int *c, int n)
{
	int i = threadIdx.x;
	if (i < n)
	{
		c[i] = a[i] + b[i];
	}
}

void s_vecAdd(int *a, int *b, int *c, int n)
{

	for (int i = 0; i < n; i++)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int *a, *b, *c;

	hipMallocManaged(&a, SIZE * sizeof(int));
	hipMallocManaged(&b, SIZE * sizeof(int));
	hipMallocManaged(&c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; i++)
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	clock_t start = clock();
	p_vecAdd<<<10, SIZE>>>(a, b, c, SIZE);
	clock_t end = clock();
	double elapsed_sec = ((double)(end - start)) / CLOCKS_PER_SEC;
	long long nanoseconds = (long long)(elapsed_sec * 1e9);
	printf("\nTime taken for kernel exexution=%lld\n nanosec", nanoseconds);

	clock_t start2 = clock();
	s_vecAdd(a, b, c, SIZE);
	clock_t end2 = clock();
	double elapsed_sec2 = (double(end2 - start2)) / CLOCKS_PER_SEC;
	long long nanosec2 = (long long)(elapsed_sec2 * 1e9);
	printf("\nTime taken for seq. execution=%lld nanosec", nanosec2);

	hipDeviceSynchronize();

	for (int i = 0; i < SIZE; i++)
	{
		printf("\nc[%d]=%d", i, c[i]);
	}
}